#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 32

inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "Cuda Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

__global__ void tiledMatrixMultKernel(float *M, float *N, float *P, int width) {
    __shared__ float M_s[TILE_WIDTH][TILE_WIDTH];
    __shared__ float N_s[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;   int by = blockIdx.y;
    int tx = threadIdx.x;  int ty = threadIdx.y;

    // Identify the row and column of the P element to work on
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    // Loop over the M and N tiles required to compute P element
    float Pvalue = 0;
    for (int tile = 0; tile < (width + TILE_WIDTH - 1) / TILE_WIDTH; tile++) {

        // Collaborative loading of M and N tiles into shared memory
        if (row < width && (tile * TILE_WIDTH + tx) < width) {
            M_s[ty][tx] = M[row * width + (tile * TILE_WIDTH + tx)];
        } else {
            M_s[ty][tx] = 0.0f;  // Zero-padding for out-of-bounds accesses
        }

        if ((tile * TILE_WIDTH + ty) < width && col < width) {
            N_s[ty][tx] = N[(tile * TILE_WIDTH + ty) * width + col];
        } else {
            N_s[ty][tx] = 0.0f;  // Zero-padding for out-of-bounds accesses
        }
        __syncthreads();

        // Compute the partial product for the tile
        for (int k = 0; k < TILE_WIDTH; k++) {
            Pvalue += M_s[ty][k] * N_s[k][tx];
        }
        __syncthreads();
    }

    // Write the result to the output matrix P
    if (row < width && col < width) {
        P[row * width + col] = Pvalue;
    }
}

void tiledMatrixMult(float *M_h, float *N_h, float *P_h, int width) {
    // Allocate GPU memory
    float *M_d, *N_d, *P_d;
    checkCuda( hipMalloc((void**)&M_d, width * width * sizeof(float)) );
    checkCuda( hipMalloc((void**)&N_d, width * width * sizeof(float)) );
    checkCuda( hipMalloc((void**)&P_d, width * width * sizeof(float)) );

    // Transfer data host -> device
    checkCuda( hipMemcpy(M_d, M_h, width * width * sizeof(float), hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(N_d, N_h, width * width * sizeof(float), hipMemcpyHostToDevice) );

    // Perform mmult
    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocks((width + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x,
                   (width + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y
    );
    tiledMatrixMultKernel<<< numBlocks, numThreadsPerBlock >>>(M_d, N_d, P_d, width);
    checkCuda( hipGetLastError() );
    checkCuda( hipDeviceSynchronize() );

    // Transfer data device -> host
    checkCuda( hipMemcpy(P_h, P_d, width * width * sizeof(float), hipMemcpyDeviceToHost) );
    
    // Free GPU memory
    checkCuda( hipFree(M_d) );
    checkCuda( hipFree(N_d) );
    checkCuda( hipFree(P_d) );
}

int main(void) {
    srand(time(NULL));

    // N x N matrix
    int n = 1 << 10;

    float *M = (float*) malloc(n * n * sizeof(float));
    float *N = (float*) malloc(n * n * sizeof(float));
    float *P = (float*) malloc(n * n * sizeof(float));

    if(!M || !N || !P) {
        fprintf(stderr, "Malloc Error.\n");
        return 1;
    }

    for (int i = 0; i < n * n; i++) {
        M[i] = rand() / (float)RAND_MAX;
        N[i] = rand() / (float)RAND_MAX;
    }

    tiledMatrixMult(M, N, P, n);

    printf("P[0] = %f | Expected:", P[0]);
    float sum = 0;
    for (int k = 0; k < n; k++) {
        sum += M[k] * N[k * n];
    }
    printf("%f\n", sum);


    free(M);
    free(N);
    free(P);

    return 0;
}