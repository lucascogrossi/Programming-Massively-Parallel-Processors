#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define TILE_DIM 32
#define COARSE_FACTOR 4

inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "Cuda Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

__global__ void tiledMatrixMultKernel(float *M, float *N, float *P, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int colStart = blockIdx.x * blockDim.x * COARSE_FACTOR + threadIdx.x;

    __shared__ float M_s[TILE_DIM][TILE_DIM];
    __shared__ float N_s[TILE_DIM][TILE_DIM];

    float sum[COARSE_FACTOR];
    for (int c = 0; c < COARSE_FACTOR; c++) {
        sum[c] = 0.0f;
    }

    for (int tile = 0; tile < width/TILE_DIM; tile++) {
        M_s[threadIdx.y][threadIdx.x] = M[row * width + tile*TILE_DIM + threadIdx.x];

        for (int c = 0; c < COARSE_FACTOR; c++) {
            int col = colStart + c*TILE_DIM;

            N_s[threadIdx.y][threadIdx.x] = N[(tile*TILE_DIM + threadIdx.y) * width + col];
            __syncthreads();

            for (int i = 0; i < TILE_DIM; i++) {
                sum[c] += M_s[threadIdx.y][i] * N_s[i][threadIdx.x];
            }
            __syncthreads();
        }
    }

    for (int c = 0; c < COARSE_FACTOR; c++) {
        int col = colStart + c*TILE_DIM;
        P[row * width + col] = sum[c];
    }

}

void tiledMatrixMult(float *M_h, float *N_h, float *P_h, int width) {
    // Allocate GPU memory
    float *M_d, *N_d, *P_d;
    checkCuda( hipMalloc((void**)&M_d, width * width * sizeof(float)) );
    checkCuda( hipMalloc((void**)&N_d, width * width * sizeof(float)) );
    checkCuda( hipMalloc((void**)&P_d, width * width * sizeof(float)) );

    // Transfer data host -> device
    checkCuda( hipMemcpy(M_d, M_h, width * width * sizeof(float), hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(N_d, N_h, width * width * sizeof(float), hipMemcpyHostToDevice) );

    // Perform mmult
    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocks((width + numThreadsPerBlock.x - 1)/COARSE_FACTOR / (numThreadsPerBlock.x ),
                   (width + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y);
    tiledMatrixMultKernel<<< numBlocks, numThreadsPerBlock >>>(M_d, N_d, P_d, width);
    checkCuda( hipGetLastError() );
    checkCuda( hipDeviceSynchronize() );

    // Transfer data device -> host
    checkCuda( hipMemcpy(P_h, P_d, width * width * sizeof(float), hipMemcpyDeviceToHost) );
    
    // Free GPU memory
    checkCuda( hipFree(M_d) );
    checkCuda( hipFree(N_d) );
    checkCuda( hipFree(P_d) );
}

int main(void) {
    srand(time(NULL));

    // N x N matrix
    int n = 1 << 12;

    float *M = (float*) malloc(n * n * sizeof(float));
    float *N = (float*) malloc(n * n * sizeof(float));
    float *P = (float*) malloc(n * n * sizeof(float));

    if(!M || !N || !P) {
        fprintf(stderr, "Malloc Error.\n");
        return 1;
    }

    for (int i = 0; i < n * n; i++) {
        M[i] = rand() / (float)RAND_MAX;
        N[i] = rand() / (float)RAND_MAX;
    }

    tiledMatrixMult(M, N, P, n);

    printf("P[0] = %f | Expected:", P[0]);
    float sum = 0;
    for (int k = 0; k < n; k++) {
        sum += M[k] * N[k * n];
    }
    printf("%f\n", sum);


    free(M);
    free(N);
    free(P);

    return 0;
}