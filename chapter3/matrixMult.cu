#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>

inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "Cuda Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

__global__ void matrixMultKernel(float *M, float *N, float *P, int width) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (col < width && row < width) {
        float Pvalue = 0;
        for (int k = 0; k < width; k++) {
            Pvalue += M[row * width + k] * N[k * width + col];
        }
        P[row * width + col] = Pvalue;
    }
}

void matrixMult(float *M_h, float *N_h, float *P_h, int width) {
    // Allocate GPU memory
    float *M_d, *N_d, *P_d;
    checkCuda( hipMalloc((void**)&M_d, width * width * sizeof(float)) );
    checkCuda( hipMalloc((void**)&N_d, width * width * sizeof(float)) );
    checkCuda( hipMalloc((void**)&P_d, width * width * sizeof(float)) );

    // Transfer data host -> device
    checkCuda( hipMemcpy(M_d, M_h, width * width * sizeof(float), hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(N_d, N_h, width * width * sizeof(float), hipMemcpyHostToDevice) );

    // Perform mmult
    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocks((width + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x,
                   (width + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y
    );
    matrixMultKernel<<< numBlocks, numThreadsPerBlock >>>(M_d, N_d, P_d, width);
    checkCuda( hipGetLastError() );
    checkCuda( hipDeviceSynchronize() );

    // Transfer data device -> host
    checkCuda( hipMemcpy(P_h, P_d, width * width * sizeof(float), hipMemcpyDeviceToHost) );
    
    // Free GPU memory
    checkCuda( hipFree(M_d) );
    checkCuda( hipFree(N_d) );
    checkCuda( hipFree(P_d) );
}

int main(void) {
    srand(time(NULL));

    // N x N matrix
    int n = 1 << 10;

    float *M = (float*) malloc(n * n * sizeof(float));
    float *N = (float*) malloc(n * n * sizeof(float));
    float *P = (float*) malloc(n * n * sizeof(float));

    if(!M || !N || !P) {
        fprintf(stderr, "Malloc Error.\n");
        return 1;
    }

    for (int i = 0; i < n * n; i++) {
        M[i] = rand() / (float)RAND_MAX;
        N[i] = rand() / (float)RAND_MAX;
    }

    matrixMult(M, N, P, n);

    printf("P[0] = %f | Expected:", P[0]);
    float sum = 0;
    for (int k = 0; k < width; k++) {
        sum += M[k] * N[k * width];
    }
    P_verify[0] = sum;
    printf("%f\n", P_verify[0]);


    free(M);
    free(N);
    free(P);

    return 0;
}